#include "hip/hip_runtime.h"
/*
 *
 *  Copyright (C) 2023, SToFU Systems S.L.
 *  All rights reserved.
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License along
 *  with this program; if not, write to the Free Software Foundation, Inc.,
 *  51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 *
 */
#include "crc32.cuh"

namespace crc32
{
    __device__ __host__ uint32_t reflect(uint32_t val, int num_bits)
    {
        uint32_t reflected = 0;
        for (int i = 0; i < num_bits; i++)
        {
            if (val & (1 << i))
                reflected |= (1 << (num_bits - 1 - i));
        }
        return reflected;
    }
    /*
    * FUNCTION: __device__ __host__ uint32_t crc32
    *
    * ARGS:
    * const uint8_t* buffer - Input buffer containing data for CRC32 calculation.
    * int size - Size of the input buffer.
    *
    * DESCRIPTION:
    * This function calculates the CRC32 checksum for a given input buffer on both CPU and GPU devices.
    * The CRC32 calculation algorithm used is the standard CRC32 polynomial with initial value of 0xFFFFFFFF and final XOR of 0xFFFFFFFF.
    * The function iterates through each byte in the input buffer using a for loop, performing bitwise XOR and shift operations to calculate the CRC32 value.
    * The calculated CRC32 value is then bitwise negated (~crc) and returned as the final result.
    *
    * RETURN VALUE:
    * uint32_t - The calculated CRC32 checksum for the input buffer.
    * This function returns a 32-bit unsigned integer representing the CRC32 checksum value.
    */
    __device__ __host__ uint32_t crc32(const uint8_t* buf, int size)
    {    
        uint32_t crcTable[256] = { \
            0x00000000, 0xF4ACFB13, 0x1DF50D35, 0xE959F626, 0x3BEA1A6A, 0xCF46E179, 0x261F175F, 0xD2B3EC4C, \
            0x77D434D4, 0x8378CFC7, 0x6A2139E1, 0x9E8DC2F2, 0x4C3E2EBE, 0xB892D5AD, 0x51CB238B, 0xA567D898, \
            0xEFA869A8, 0x1B0492BB, 0xF25D649D, 0x06F19F8E, 0xD44273C2, 0x20EE88D1, 0xC9B77EF7, 0x3D1B85E4, \
            0x987C5D7C, 0x6CD0A66F, 0x85895049, 0x7125AB5A, 0xA3964716, 0x573ABC05, 0xBE634A23, 0x4ACFB130, \
            0x2BFC2843, 0xDF50D350, 0x36092576, 0xC2A5DE65, 0x10163229, 0xE4BAC93A, 0x0DE33F1C, 0xF94FC40F, \
            0x5C281C97, 0xA884E784, 0x41DD11A2, 0xB571EAB1, 0x67C206FD, 0x936EFDEE, 0x7A370BC8, 0x8E9BF0DB, \
            0xC45441EB, 0x30F8BAF8, 0xD9A14CDE, 0x2D0DB7CD, 0xFFBE5B81, 0x0B12A092, 0xE24B56B4, 0x16E7ADA7, \
            0xB380753F, 0x472C8E2C, 0xAE75780A, 0x5AD98319, 0x886A6F55, 0x7CC69446, 0x959F6260, 0x61339973, \
            0x57F85086, 0xA354AB95, 0x4A0D5DB3, 0xBEA1A6A0, 0x6C124AEC, 0x98BEB1FF, 0x71E747D9, 0x854BBCCA, \
            0x202C6452, 0xD4809F41, 0x3DD96967, 0xC9759274, 0x1BC67E38, 0xEF6A852B, 0x0633730D, 0xF29F881E, \
            0xB850392E, 0x4CFCC23D, 0xA5A5341B, 0x5109CF08, 0x83BA2344, 0x7716D857, 0x9E4F2E71, 0x6AE3D562, \
            0xCF840DFA, 0x3B28F6E9, 0xD27100CF, 0x26DDFBDC, 0xF46E1790, 0x00C2EC83, 0xE99B1AA5, 0x1D37E1B6, \
            0x7C0478C5, 0x88A883D6, 0x61F175F0, 0x955D8EE3, 0x47EE62AF, 0xB34299BC, 0x5A1B6F9A, 0xAEB79489, \
            0x0BD04C11, 0xFF7CB702, 0x16254124, 0xE289BA37, 0x303A567B, 0xC496AD68, 0x2DCF5B4E, 0xD963A05D, \
            0x93AC116D, 0x6700EA7E, 0x8E591C58, 0x7AF5E74B, 0xA8460B07, 0x5CEAF014, 0xB5B30632, 0x411FFD21, \
            0xE47825B9, 0x10D4DEAA, 0xF98D288C, 0x0D21D39F, 0xDF923FD3, 0x2B3EC4C0, 0xC26732E6, 0x36CBC9F5, \
            0xAFF0A10C, 0x5B5C5A1F, 0xB205AC39, 0x46A9572A, 0x941ABB66, 0x60B64075, 0x89EFB653, 0x7D434D40, \
            0xD82495D8, 0x2C886ECB, 0xC5D198ED, 0x317D63FE, 0xE3CE8FB2, 0x176274A1, 0xFE3B8287, 0x0A977994, \
            0x4058C8A4, 0xB4F433B7, 0x5DADC591, 0xA9013E82, 0x7BB2D2CE, 0x8F1E29DD, 0x6647DFFB, 0x92EB24E8, \
            0x378CFC70, 0xC3200763, 0x2A79F145, 0xDED50A56, 0x0C66E61A, 0xF8CA1D09, 0x1193EB2F, 0xE53F103C, \
            0x840C894F, 0x70A0725C, 0x99F9847A, 0x6D557F69, 0xBFE69325, 0x4B4A6836, 0xA2139E10, 0x56BF6503, \
            0xF3D8BD9B, 0x07744688, 0xEE2DB0AE, 0x1A814BBD, 0xC832A7F1, 0x3C9E5CE2, 0xD5C7AAC4, 0x216B51D7, \
            0x6BA4E0E7, 0x9F081BF4, 0x7651EDD2, 0x82FD16C1, 0x504EFA8D, 0xA4E2019E, 0x4DBBF7B8, 0xB9170CAB, \
            0x1C70D433, 0xE8DC2F20, 0x0185D906, 0xF5292215, 0x279ACE59, 0xD336354A, 0x3A6FC36C, 0xCEC3387F, \
            0xF808F18A, 0x0CA40A99, 0xE5FDFCBF, 0x115107AC, 0xC3E2EBE0, 0x374E10F3, 0xDE17E6D5, 0x2ABB1DC6, \
            0x8FDCC55E, 0x7B703E4D, 0x9229C86B, 0x66853378, 0xB436DF34, 0x409A2427, 0xA9C3D201, 0x5D6F2912, \
            0x17A09822, 0xE30C6331, 0x0A559517, 0xFEF96E04, 0x2C4A8248, 0xD8E6795B, 0x31BF8F7D, 0xC513746E, \
            0x6074ACF6, 0x94D857E5, 0x7D81A1C3, 0x892D5AD0, 0x5B9EB69C, 0xAF324D8F, 0x466BBBA9, 0xB2C740BA, \
            0xD3F4D9C9, 0x275822DA, 0xCE01D4FC, 0x3AAD2FEF, 0xE81EC3A3, 0x1CB238B0, 0xF5EBCE96, 0x01473585, \
            0xA420ED1D, 0x508C160E, 0xB9D5E028, 0x4D791B3B, 0x9FCAF777, 0x6B660C64, 0x823FFA42, 0x76930151, \
            0x3C5CB061, 0xC8F04B72, 0x21A9BD54, 0xD5054647, 0x07B6AA0B, 0xF31A5118, 0x1A43A73E, 0xEEEF5C2D, \
            0x4B8884B5, 0xBF247FA6, 0x567D8980, 0xA2D17293, 0x70629EDF, 0x84CE65CC, 0x6D9793EA, 0x993B68F9 \
        };

        uint32_t crc = ~0U;
        while(size--){
            uint8_t byte = buf[size];
            uint32_t idx = ((crc >> 24) ^ reflect(byte, 8)) & 0xFF;
            crc = crcTable[idx] ^ (crc << 8);
        }

        crc = reflect(crc, 32);

        // invert the crc value
            // easiest way is to xor 0xffff....
        return crc ^ ~0U;
    }
    /*
    * FUNCTION: __global__ void crc32Kernel
    *
    * ARGS:
    * In uint8_t* buffers - Input buffer containing data for CRC32 calculation.
    * Out uint32_t* crcResults - Output buffer to store CRC32 results.
    * int numBuffers - Number of input buffers.
    * int bufferSize - Size of each input buffer.
    *
    * DESCRIPTION:
    * This is a CUDA kernel function for calculating CRC32 checksums in parallel on a GPU device.
    * Each thread in the GPU grid corresponds to a unique thread identifier (tid) calculated from blockIdx.x and blockDim.x.
    * The bufferIndex is calculated based on tid and bufferSize to determine the starting index of the current buffer to be processed.
    * The function performs CRC32 calculation on each buffer by iterating through each byte in the buffer using a for loop.
    * The calculated CRC32 value is then saved to the crcResults array at the corresponding tid index.
    *
    * RETURN VALUE: void
    * This function does not return a value.
    */
    __global__ void crc32Kernel(_In_ const uint8_t* buffers, _Out_ uint32_t* crcResults, int numBuffers, int bufferSize)
    {
        
        /* Calculate unique thread identifier */
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        /* Calculate index of the current buffer */
        int bufferIndex = tid * bufferSize;

        /* Check if buffer index is within valid range */
        if (bufferIndex < numBuffers * bufferSize)
            /* Call crc32 function to calculate CRC32 for the current buffer */
            crcResults[tid] = crc32(buffers + bufferIndex, bufferSize);
    }
}