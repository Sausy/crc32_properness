#include "hip/hip_runtime.h"
/*
 *
 *  Copyright (C) 2023, SToFU Systems S.L.
 *  All rights reserved.
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License along
 *  with this program; if not, write to the Free Software Foundation, Inc.,
 *  51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 *
 */
#include "sha512.cuh"

namespace hashes
{
    __device__ static const uint64_t K[80] = {
   UINT64_C(0x428a2f98d728ae22), UINT64_C(0x7137449123ef65cd),
   UINT64_C(0xb5c0fbcfec4d3b2f), UINT64_C(0xe9b5dba58189dbbc),
   UINT64_C(0x3956c25bf348b538), UINT64_C(0x59f111f1b605d019),
   UINT64_C(0x923f82a4af194f9b), UINT64_C(0xab1c5ed5da6d8118),
   UINT64_C(0xd807aa98a3030242), UINT64_C(0x12835b0145706fbe),
   UINT64_C(0x243185be4ee4b28c), UINT64_C(0x550c7dc3d5ffb4e2),
   UINT64_C(0x72be5d74f27b896f), UINT64_C(0x80deb1fe3b1696b1),
   UINT64_C(0x9bdc06a725c71235), UINT64_C(0xc19bf174cf692694),
   UINT64_C(0xe49b69c19ef14ad2), UINT64_C(0xefbe4786384f25e3),
   UINT64_C(0x0fc19dc68b8cd5b5), UINT64_C(0x240ca1cc77ac9c65),
   UINT64_C(0x2de92c6f592b0275), UINT64_C(0x4a7484aa6ea6e483),
   UINT64_C(0x5cb0a9dcbd41fbd4), UINT64_C(0x76f988da831153b5),
   UINT64_C(0x983e5152ee66dfab), UINT64_C(0xa831c66d2db43210),
   UINT64_C(0xb00327c898fb213f), UINT64_C(0xbf597fc7beef0ee4),
   UINT64_C(0xc6e00bf33da88fc2), UINT64_C(0xd5a79147930aa725),
   UINT64_C(0x06ca6351e003826f), UINT64_C(0x142929670a0e6e70),
   UINT64_C(0x27b70a8546d22ffc), UINT64_C(0x2e1b21385c26c926),
   UINT64_C(0x4d2c6dfc5ac42aed), UINT64_C(0x53380d139d95b3df),
   UINT64_C(0x650a73548baf63de), UINT64_C(0x766a0abb3c77b2a8),
   UINT64_C(0x81c2c92e47edaee6), UINT64_C(0x92722c851482353b),
   UINT64_C(0xa2bfe8a14cf10364), UINT64_C(0xa81a664bbc423001),
   UINT64_C(0xc24b8b70d0f89791), UINT64_C(0xc76c51a30654be30),
   UINT64_C(0xd192e819d6ef5218), UINT64_C(0xd69906245565a910),
   UINT64_C(0xf40e35855771202a), UINT64_C(0x106aa07032bbd1b8),
   UINT64_C(0x19a4c116b8d2d0c8), UINT64_C(0x1e376c085141ab53),
   UINT64_C(0x2748774cdf8eeb99), UINT64_C(0x34b0bcb5e19b48a8),
   UINT64_C(0x391c0cb3c5c95a63), UINT64_C(0x4ed8aa4ae3418acb),
   UINT64_C(0x5b9cca4f7763e373), UINT64_C(0x682e6ff3d6b2b8a3),
   UINT64_C(0x748f82ee5defb2fc), UINT64_C(0x78a5636f43172f60),
   UINT64_C(0x84c87814a1f0ab72), UINT64_C(0x8cc702081a6439ec),
   UINT64_C(0x90befffa23631e28), UINT64_C(0xa4506cebde82bde9),
   UINT64_C(0xbef9a3f7b2c67915), UINT64_C(0xc67178f2e372532b),
   UINT64_C(0xca273eceea26619c), UINT64_C(0xd186b8c721c0c207),
   UINT64_C(0xeada7dd6cde0eb1e), UINT64_C(0xf57d4f7fee6ed178),
   UINT64_C(0x06f067aa72176fba), UINT64_C(0x0a637dc5a2c898a6),
   UINT64_C(0x113f9804bef90dae), UINT64_C(0x1b710b35131c471b),
   UINT64_C(0x28db77f523047d84), UINT64_C(0x32caab7b40c72493),
   UINT64_C(0x3c9ebe0a15c9bebc), UINT64_C(0x431d67c49c100d4c),
   UINT64_C(0x4cc5d4becb3e42b6), UINT64_C(0x597f299cfc657e2a),
   UINT64_C(0x5fcb6fab3ad6faec), UINT64_C(0x6c44198c4a475817)
    };

    /* Various logical functions for calculating sha-512 hash on GPU */

#define ROR64c(x, y) \
    ( ((((x)&UINT64_C(0xFFFFFFFFFFFFFFFF))>>((uint64_t)(y)&UINT64_C(63))) | \
      ((x)<<((uint64_t)(64-((y)&UINT64_C(63)))))) & UINT64_C(0xFFFFFFFFFFFFFFFF))

#define STORE64H(x, y)                                                                     \
   { (y)[0] = (unsigned char)(((x)>>56)&255); (y)[1] = (unsigned char)(((x)>>48)&255);     \
     (y)[2] = (unsigned char)(((x)>>40)&255); (y)[3] = (unsigned char)(((x)>>32)&255);     \
     (y)[4] = (unsigned char)(((x)>>24)&255); (y)[5] = (unsigned char)(((x)>>16)&255);     \
     (y)[6] = (unsigned char)(((x)>>8)&255); (y)[7] = (unsigned char)((x)&255); }

#define LOAD64H(x, y)                                                      \
   { x = (((uint64_t)((y)[0] & 255))<<56)|(((uint64_t)((y)[1] & 255))<<48) | \
         (((uint64_t)((y)[2] & 255))<<40)|(((uint64_t)((y)[3] & 255))<<32) | \
         (((uint64_t)((y)[4] & 255))<<24)|(((uint64_t)((y)[5] & 255))<<16) | \
         (((uint64_t)((y)[6] & 255))<<8)|(((uint64_t)((y)[7] & 255))); }


#define Ch(x,y,z)       (z ^ (x & (y ^ z)))
#define Maj(x,y,z)      (((x | y) & z) | (x & y))
#define S(x, n)         ROR64c(x, n)
#define R(x, n)         (((x) &UINT64_C(0xFFFFFFFFFFFFFFFF))>>((uint64_t)n))
#define Sigma0(x)       (S(x, 28) ^ S(x, 34) ^ S(x, 39))
#define Sigma1(x)       (S(x, 14) ^ S(x, 18) ^ S(x, 41))
#define Gamma0(x)       (S(x, 1) ^ S(x, 8) ^ R(x, 7))
#define Gamma1(x)       (S(x, 19) ^ S(x, 61) ^ R(x, 6))
#ifndef MIN
#define MIN(x, y) ( ((x)<(y))?(x):(y) )
#endif

    /*
    * FUNCTION: static int __device__ __host__ sha512_compress
    *
    * ARGS:
    * sha512_context* md - Pointer to the SHA-512 context structure.
    * unsigned char* buf - Pointer to the buffer containing the data to be compressed.
    *
    * DESCRIPTION:
    * This function performs the compression step of the SHA-512 algorithm on a block of data.
    * It performs the following steps:
    * - Copies the current state values from the SHA-512 context (md) into local variables (S).
    * - Copies the input data block (buf) into an array of 80 64-bit unsigned integers (W).
    * - Fills the remaining elements of W[16..79] using bitwise operations and additions as per the SHA-512 algorithm.
    * - Performs a series of 80 rounds of SHA-512 operations (RND macro) on the state variables (S) and elements of W.
    * - Updates the state variables (md->state) by adding the values from the local variables (S).
    * This function is marked as static, which means it can only be accessed within the same source file. It can be called from both device (GPU) and host (CPU) code, as denoted by the __device__ and __host__ qualifiers.
    *
    * RETURN VALUE: int
    * Returns 0 on success, and a non-zero value if any error occurs (currently not used in the function).
    */
    static int __device__ __host__ sha512_compress(sha512_context* md, unsigned char* buf)
    {
        uint64_t S[8], W[80], t0, t1;
        int i;

        /* copy state into S */
        for (i = 0; i < 8; i++)
            S[i] = md->state[i];
        /* copy the state into 1024-bits into W[0..15] */
        for (i = 0; i < 16; i++)
            LOAD64H(W[i], buf + (8 * i));
        /* fill W[16..79] */
        for (i = 16; i < 80; i++)
            W[i] = Gamma1(W[i - 2]) + W[i - 7] + Gamma0(W[i - 15]) + W[i - 16];

        /* Compress */
#define RND(a,b,c,d,e,f,g,h,i) \
    t0 = h + Sigma1(e) + Ch(e, f, g) + K[i] + W[i]; \
    t1 = Sigma0(a) + Maj(a, b, c);\
    d += t0; \
    h  = t0 + t1;

        for (i = 0; i < 80; i += 8) {
            RND(S[0], S[1], S[2], S[3], S[4], S[5], S[6], S[7], i + 0);
            RND(S[7], S[0], S[1], S[2], S[3], S[4], S[5], S[6], i + 1);
            RND(S[6], S[7], S[0], S[1], S[2], S[3], S[4], S[5], i + 2);
            RND(S[5], S[6], S[7], S[0], S[1], S[2], S[3], S[4], i + 3);
            RND(S[4], S[5], S[6], S[7], S[0], S[1], S[2], S[3], i + 4);
            RND(S[3], S[4], S[5], S[6], S[7], S[0], S[1], S[2], i + 5);
            RND(S[2], S[3], S[4], S[5], S[6], S[7], S[0], S[1], i + 6);
            RND(S[1], S[2], S[3], S[4], S[5], S[6], S[7], S[0], i + 7);
        }
#undef RND
        for (i = 0; i < 8; i++)
            md->state[i] = md->state[i] + S[i];

        return 0;
    }

    /*
    * FUNCTION: int __device__ __host__ sha512_init
    *
    * ARGS:
    * sha512_context* md - Pointer to the SHA-512 context structure.
    *
    * DESCRIPTION:
    * This function initializes the SHA-512 context by setting the initial state values for the SHA-512 hash calculation.
    * It performs the following steps:
    * - Checks for a NULL pointer for the input SHA-512 context, which is an error condition.
    * - Sets the buffer length (curlen) and original message length (length) in the context to 0.
    * - Sets the initial state values (8 64-bit unsigned integers) in the context as per the SHA-512 algorithm specifications.
    * This function can be called from both device (GPU) and host (CPU) code, as denoted by the __device__ and __host__ qualifiers.
    *
    * RETURN VALUE: int
    * Returns 0 on success, and a non-zero value if any error occurs (e.g., NULL pointer for the input context).
    */
    int __device__ __host__ sha512_init(sha512_context* md)
    {
        if (md == NULL) return 1;
        md->curlen = 0;
        md->length = 0;
        md->state[0] = UINT64_C(0x6a09e667f3bcc908);
        md->state[1] = UINT64_C(0xbb67ae8584caa73b);
        md->state[2] = UINT64_C(0x3c6ef372fe94f82b);
        md->state[3] = UINT64_C(0xa54ff53a5f1d36f1);
        md->state[4] = UINT64_C(0x510e527fade682d1);
        md->state[5] = UINT64_C(0x9b05688c2b3e6c1f);
        md->state[6] = UINT64_C(0x1f83d9abfb41bd6b);
        md->state[7] = UINT64_C(0x5be0cd19137e2179);

        return 0;
    }

    /*
    * FUNCTION: int __device__ __host__ sha512_update
    *
    * ARGS:
    * sha512_context* md - Pointer to the SHA-512 context structure.
    * const uint8_t* in - Pointer to the input message buffer.
    * size_t inlen - Length of the input message buffer.
    *
    * DESCRIPTION:
    * This function updates the SHA-512 hash calculation with additional input data. It processes the input data in blocks of 128 bytes and updates the SHA-512 context accordingly.
    * It performs the following steps:
    * - Checks for NULL pointers for the input SHA-512 context and input message buffer.
    * - Checks if the current length of the message buffer in the context is greater than the size of the buffer, which is an error condition.
    * - Processes the input data in blocks of 128 bytes:
    * - If the current length of the message buffer in the context is 0 and the input data length is greater than or equal to 128 bytes, it directly compresses the input data using sha512_compress() function, updates the length of the original message, and advances the input data buffer and length.
    * - Otherwise, it copies the input data to the message buffer in the context until the buffer is full (128 bytes):
    * - If the buffer is full, it compresses the buffer using sha512_compress() function, updates the length of the original message, and resets the buffer length.
    * - Continues this process until all the input data is processed.
    * This function can be called from both device (GPU) and host (CPU) code, as denoted by the __device__ and __host__ qualifiers.
    *
    * RETURN VALUE: int
    * Returns 0 on success, and a non-zero value if any error occurs.
    */
    int __device__ __host__ sha512_update(sha512_context* md, const uint8_t* in, size_t inlen)
    {
        size_t n;
        int  err;

        /* Check if input parameters are valid */
        if (md == NULL) return 1;
        if (in == NULL) return 1;
        if (md->curlen > sizeof(md->buf)) return 1;

        /* Process input data in blocks of HASH_SIZE bytes */
        while (inlen > 0)
        {
            /* If there is enough input data and buffer is empty, directly compress the input data */
            if (md->curlen == 0 && inlen >= HASH_SIZE)
            {
                if ((err = sha512_compress(md, (unsigned char*)in)) != 0) return err;

                md->length += HASH_SIZE * 8;
                in += HASH_SIZE;
                inlen -= HASH_SIZE;
            }
            else
            {
                /* Copy input data to buffer until it is full or input data is exhausted */
                n = MIN(inlen, (HASH_SIZE - md->curlen));
                for (size_t i = 0; i < n; ++i)
                    md->buf[i + md->curlen] = in[i];

                md->curlen += n;
                in += n;
                inlen -= n;

                /* If buffer is full, compress it */
                if (md->curlen == HASH_SIZE) {
                    if ((err = sha512_compress(md, md->buf)) != 0) return err;

                    md->length += 8 * HASH_SIZE;
                    md->curlen = 0;
                }
            }
        }
        return 0;
    }

    /*
    * FUNCTION: int __device__ __host__ sha512_final
    *
    * ARGS:
    * sha512_context* md - Pointer to the SHA-512 context structure.
    * uint8_t* out - Pointer to the output buffer for storing the final SHA-512 hash.
    *
    * DESCRIPTION:
    * This function finalizes the SHA-512 hash calculation by padding the input message and storing the calculated hash in the output buffer.
    * It performs the following steps:
    * - Checks for NULL pointers for the input SHA-512 context and output buffer.
    * - Appends the '1' bit to the message buffer.
    * - If the length of the message buffer is greater than 112 bytes, it appends zeros and compresses the buffer.
    * - Appends zeros to the message buffer until it reaches a length of 120 bytes.
    * - Stores the length of the original message in big-endian format in the last 8 bytes of the buffer.
    * - Performs the final compression using sha512_compress() function.
    * - Copies the resulting hash from the SHA-512 context to the output buffer.
    *  This function can be called from both device (GPU) and host (CPU) code, as denoted by the __device__ and __host__ qualifiers.
    *
    * RETURN VALUE: int
    * Returns 0 on success, and a non-zero value if any error occurs.
    */
    int __device__ __host__ sha512_final(sha512_context* md, uint8_t* out)
    {
        /* Check if input parameters are valid */
        if (md == NULL) return 1;
        if (out == NULL) return 1;
        if (md->curlen >= sizeof(md->buf)) return 1;

        /* increase the length of the message */
        md->length += md->curlen * UINT64_C(8);
        /* append the '1' bit */
        md->buf[md->curlen++] = (unsigned char)0x80;

        /* if the length is currently above 112 bytes append zeros then compress. Then can fall back to padding zeros and length encoding like normal */
        if (md->curlen > 112) {
            while (md->curlen < HASH_SIZE)
                md->buf[md->curlen++] = (unsigned char)0;

            sha512_compress(md, md->buf);
            md->curlen = 0;
        }

        while (md->curlen < 120)
            md->buf[md->curlen++] = (unsigned char)0;

        /* store length */
        STORE64H(md->length, md->buf + 120);
        sha512_compress(md, md->buf);
        /* copy output */
        for (int i = 0; i < 8; i++)
            STORE64H(md->state[i], out + (8 * i));

        return 0;
    }

    /*
    * FUNCTION: int __device__ __host__ sha512
    *
    * ARGS:
    * const uint8_t* message - Pointer to the input message whose SHA-512 hash needs to be calculated.
    * size_t length - Length of the input message.
    * uint8_t* out - Pointer to the output buffer for storing the calculated SHA-512 hash.
    *
    * DESCRIPTION:
    * This function calculates the SHA-512 hash for the input message using the sha512_context structure and associated functions.
    * It initializes the sha512_context using sha512_init() function, updates the context with the input message using sha512_update() function, and finalizes the context to obtain the SHA-512 hash using sha512_final() function.
    * The calculated hash is stored in the output buffer pointed to by 'out'.
    * This function can be called from both device (GPU) and host (CPU) code, as denoted by the __device__ and __host__ qualifiers.
    *
    * RETURN VALUE: int
    * Returns the status of the SHA-512 calculation, where 0 indicates success, and any other value indicates an error.
    */
    int __device__ __host__ sha512(const uint8_t* message, size_t length, uint8_t* out)
    {
        sha512_context ctx;
        int status;
        if ((status = sha512_init(&ctx))) return status;
        if ((status = sha512_update(&ctx, message, length))) return status;
        if ((status = sha512_final(&ctx, out))) return status;
        return status;
    }

    /*
    * FUNCTION: std::string __host__ sha512
    *
    * ARGS:
    * const uint8_t* message - Pointer to the input message whose SHA-512 hash needs to be calculated.
    * size_t length - Length of the input data.
    *
    * DESCRIPTION:
    * This function calculates the SHA-512 hash of the input data using a GPU-based implementation.
    * It performs the following steps:
    * - Initializes a SHA-512 context structure (ctx) from the sha512GPU namespace.
    * - Updates the context with the input data using sha512GPU::sha512_update() function.
    * - Finalizes the hash calculation and stores the resulting digest in a local buffer (digest) using sha512GPU::sha512_final() function.
    * - Converts the digest from binary to hexadecimal representation and stores it in a string buffer (buf) using sprintf() function.
    * - Returns the calculated SHA-512 hash as a string.
    * This function is marked with __host__ qualifier, which means it can be called from host (CPU) code, but not from device (GPU) code.
    *
    * RETURN VALUE: std::vector<uint8_t>
    * Returns the calculated std::vector<uint8_t> as a hexadecimal bytes.
    */
    std::vector<uint8_t> __host__ sha512(const uint8_t* message, size_t length)
    {
        std::vector<uint8_t> digest(DIGEST_SIZE);
        hashes::sha512_context ctx;
        int status;
        if ((status = hashes::sha512_init(&ctx))) return digest;
        if ((status = hashes::sha512_update(&ctx, message, length))) return digest;
        if ((status = hashes::sha512_final(&ctx, digest.data()))) return digest;
        return digest;
    }

    /*
    * FUNCTION: void __global__ sha512Kernel
    *
    * ARGS:
    * char* inputs - Pointer to the input buffers in GPU memory.
    * int numInputs - Number of input buffers to process.
    * uint8_t* outputs - Pointer to the output buffer in GPU memory for storing the calculated SHA-512 hashes.
    * size_t bufferSize - Size of each input buffer.
    * int bufferLength - Length of each input buffer.
    * This function is meant to be called from host code and executed on the GPU using CUDA.
    *
    * DESCRIPTION:
    * This CUDA kernel function is launched on the GPU to calculate the SHA-512 hashes for the input buffers in parallel.
    * It calculates the global thread ID using blockIdx.x and threadIdx.x, and checks if the thread ID is within bounds of the number of input buffers.
    * If the thread ID is within bounds, it calls the sha512() function to calculate the SHA-512 hash for the corresponding input buffer, and stores the result in the output buffer in GPU memory.
    */
    void __global__ sha512Kernel(char* inputs, int numInputs, uint8_t* outputs, int bufferLength)
    {
        /* Calculate global thread ID */
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        /* Check if thread ID is within bounds and call SHA-512 function */
        if (index < numInputs)
            sha512((uint8_t*)(inputs + index * bufferLength), bufferLength, outputs + index * DIGEST_SIZE);
    }
}